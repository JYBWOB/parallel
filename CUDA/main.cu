#include<iostream>
#include<vector>
#include<cmath>
#include<stdio.h>
#include<cstring>
#include<time.h>
#include<fstream>
#include <stdlib.h>
#include "hip/hip_runtime.h"

using namespace std;

#define BLOCK_SIZE 32
#define BLOCK_LENGTH 128

const int maxN = 2048;
//A为一个N*N的矩阵的对称矩阵
float *A;
//数组b
float *b;
//残差r,结果x,计算方向向量d
float *r;
float *d;
float *x;
float *dtAdMatrix;
float *dtAdVector;


// int MAX_ITER_TIME = 5000;
// bool FIX_ITER_TIME = true;
int MAX_ITER_TIME = 500000;
bool FIX_ITER_TIME = false;

struct timespec sts, ets;

void displayMatrix(float *a, int N){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            cout<<a[i*N+j]<<"    ";
        }
        cout<<endl;
    }
}

void displayVector(float *b, int N){
    for(int i=0;i< N;i++){
        cout<<b[i]<<" ";
    }
    cout<<endl;
}

//计算内积
float INNER_PRODUCT(float *a, float *b, int N){
    float res = 0.0;
    for(int i = 0; i < N; i++) {
        res += a[i]*b[i];
    }
    return res;
    // __shared__ float res = 0;
    // int row = blockIdx.y * blockDim.y + threadIdx.y;
    // int col = blockIdx.x * blockDim.x + threadIdx.x;
    // if(row == 0 && col < N) {
    //     res += a[col] * b[col];
    // }
    // return res;
}

//更新残差 r = A*x-b
void  MATRIX_VECTOR_PRODUCT(float *r, float *a, float *x,float *b, int N){
    float temp = 0;
    for(int i=0;i<N;i++){
        temp = 0;
        for(int j=0;j<N;j++){
            temp += a[i * N + j]*x[j];
        }
        r[i] = temp - b[i];
    }
}

//计算 d转置 * a矩阵 * d
__global__ void MATRIX_PRODUCT(float* res, float *a, float *d, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("cuda(%d, %d): %f\n", row, col, a[row * N + col]);
    if(col < N && row < N)
        res[row * N + col] = d[row] * a[row * N + col] * d[col];
}

// 求和
__global__ void SUM_MATRIX(float* res, float *a, int N){
    extern __shared__ float sdata[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // 一个线程负责把一个元素从全局内存载入到共享内存
    float x = 0;
    if (i < N)
    {
        x = a[i];
    }
    sdata[threadIdx.x] = x;
    __syncthreads();// 等待所有线程把自己负责的元素载入到共享内存
    // 块内进行合并操作，每次合并变为一半
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (threadIdx.x < offset)// 控制只有某些线程才进行操作。
        {
            // add a partial sum upstream to our own
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        // wait until all threads in the block have updated their partial sums
        __syncthreads();
    }
    // 每个块的线程0负责存放块内求和的结果
    if (threadIdx.x == 0)
    {
        res[blockIdx.x] = sdata[0];
    }
}

void MATRIX_PRODUCT_CPU(float* res, float *a, float *d, int N){
    *res = 0.0;
    for(int i=0;i<N;i++){
        for(int j = 0;j<N;j++){
            *res += d[i]*a[i*N+j]*d[j];
        }
    }
}


int main(){
    hipMallocManaged((void **) &A, sizeof(float)*maxN*maxN);
    hipMallocManaged((void **) &b, sizeof(float)*maxN);
    hipMallocManaged((void **) &r, sizeof(float)*maxN);
    hipMallocManaged((void **) &d, sizeof(float)*maxN);
    hipMallocManaged((void **) &x, sizeof(float)*maxN);
    hipMallocManaged((void **) &dtAdMatrix, sizeof(float)*maxN*maxN);
    hipMallocManaged((void **) &dtAdVector, sizeof(float)*maxN);
    fstream file("res_base.csv", ofstream::out);
    for(int N = 128; N <= maxN; N += 128) {
    // for(int N = 4; N <= 1024; N += 4) {
        //初始化A
        for(int i=0;i<N;i++){
            for(int j =0;j<N;j++){
                if(i==j){
                  A[i * N + j] = 2;
                }
                else if(abs(i-j) == 1){
                  A[i * N + j] = -1;
                }
                else {
                  A[i * N + j] = 0;
                }
            }
        }
        for(int i = 0; i < N; i++) {
            b[i] = 1.0;
            x[i] = 0;
        }
        MATRIX_VECTOR_PRODUCT(r, A, x, b, N);
        for(int i = 0; i < N; i++) {
            d[i] = -r[i];
        }
        // displayMatrix(A, N);
        // displayVector(b, N);
        // displayVector(x, N);
        // displayVector(r, N);
        // displayVector(d, N);
        
        //displayMatrix(A, N);
        //displayVector(b, N);
        // cudaMemcpy(d_A, h_A, sizeof(float)*N*N, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_b, h_b, sizeof(float)*N, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_r, h_r, sizeof(float)*N, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_d, h_d, sizeof(float)*N, cudaMemcpyHostToDevice);
        // cudaMemcpy(d_x, h_x, sizeof(float)*N, cudaMemcpyHostToDevice);

        unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

        int count = 0;
        timespec_get(&sts, TIME_UTC);
        //开始迭代
        for(int i =0;i<MAX_ITER_TIME;i++){
            count++;
            float r2 = INNER_PRODUCT(r, r, N);
            // displayMatrix(A, N);
            // displayVector(d, N);

            MATRIX_PRODUCT<<<dimGrid, dimBlock>>>(dtAdMatrix, A, d, N);
            hipDeviceSynchronize();
            int gridSize = (N * N + BLOCK_LENGTH - 1) / BLOCK_LENGTH;
            SUM_MATRIX<<<gridSize, BLOCK_LENGTH>>>(dtAdVector, dtAdMatrix, N * N);
            hipError_t cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess) 
            {
                fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            }
            hipDeviceSynchronize();
            float dtAd = 0.0;
            for(int p = 0; p < gridSize; p++) {
                dtAd += dtAdVector[p];
            }

            // cout << "cuda:" << dtAd << endl;
            // float ppp;
            // MATRIX_PRODUCT_CPU(&ppp, A, d, N);
            // cout << "cpu:" << ppp << endl;
            // cin >> ppp;

            // float ppp;
            // MATRIX_PRODUCT_CPU(&dtAd, A, d, N);
            // cout << "cpu:" << dtAd << endl;
            // cin >> ppp;


            //计算步长
            float alpha = r2/dtAd;
            //修正x
            for(int j=0;j<N;j++){
                x[j] = x[j] + alpha*d[j];
                r[j] = r[j] + alpha*INNER_PRODUCT(&A[j*N], d, N);
            }
            float r2n = INNER_PRODUCT(r, r, N);
            if(!FIX_ITER_TIME && r2n < 1e-4)
                break;
            int beta = r2n / r2;
            for(int j=0; j < N; j++) {
                d[j] = -r[j] + beta * d[j];
            }
        }
        timespec_get(&ets, TIME_UTC);
        time_t dsec = ets.tv_sec - sts.tv_sec;
        unsigned long long dnsec = ets.tv_nsec - sts.tv_nsec;
        cout << N << "：\t" << dsec << "." << dnsec << "\t" << count << endl;
        file << N << "," << dsec << "." << dnsec << "," << count << endl;
        // displayVector(x, N);
    }
    file.close();
    return 0;
}
